#include "hip/hip_runtime.h"
// Estimator CUDA wrapper
// 
// photonpy - Single molecule localization microscopy library
// © Jelmer Cnossen 2018-2021
#include "EstimatorImpl.h"
#include "CudaUtils.h"
#include <unordered_map>

// For ComputeCOM
#include "Gaussian/GaussianPSFModels.h"

CDLL_EXPORT Estimator* Estimator_WrapCUDA(cuEstimator* psf)
{
	return new cuEstimatorWrapper(psf);
}



cuEstimator::DeviceBuffers::DeviceBuffers(int smpcount, int numspots, int k)
	: numspots(numspots), 
	psf_deriv(smpcount*k*numspots), 
	psf_ev(smpcount*numspots), 
	lm_alphabeta(numspots*k*(k+1)), 
	lm_lu(numspots*k*k),
	invert_temp(numspots*(k+1))
{}

cuEstimator::DeviceBuffers::~DeviceBuffers() {}

cuEstimator::DeviceBuffers* cuEstimator::GetDeviceBuffers(hipStream_t stream, int numspots)
{
	return LockedFunction(streamDataMutex, [&]() {
		auto it = streamData.find(stream);

		if (it != streamData.end() && it->second.numspots < numspots) {
			streamData.erase(it);
			it = streamData.end();
		}

		if (it == streamData.end())
			it = streamData.emplace(stream, DeviceBuffers(SampleCount(), numspots, NumParams())).first;
		return &it->second;
		});
}


cuEstimator::cuEstimator(const std::vector<int>& sampleSize, int numConst, int diagsize, 
	const char* paramFormat, std::vector<ParamLimit> limits) : 
	EstimatorBase(sampleSize, numConst, diagsize, paramFormat, limits),
	d_limits(limits)
{}

cuEstimator::~cuEstimator()
{}


void cuEstimator::ChiSquareAndCRLB(const float* d_params, const float* sample, const float* d_const, 
	const int* d_roipos,  float* crlb, float* chisq, int numspots, hipStream_t stream)
{
	auto buffers = GetDeviceBuffers(stream, numspots);

	float* d_deriv = buffers->psf_deriv.data();
	float* d_ev = buffers->psf_ev.data();
	float* d_fi = buffers->lm_lu.data(); // happens to be also K*K elements
	float* d_fi_inv = buffers->lm_alphabeta.data();
	int* d_P = buffers->invert_temp.data();

	Derivatives(d_deriv, d_ev, d_params, d_const, d_roipos, numspots, stream);

	int K = NumParams();
	int smpcount = SampleCount();

	LaunchKernel(numspots, [=]__device__(int spot) {
		float* spot_crlb = &crlb[K * spot];
		float* fi = &d_fi[K*K*spot];
		for (int i = 0; i < K*K; i++)
			fi[i] = 0;

		float spot_chisq = 0.0f;
		const float *spot_deriv = &d_deriv[spot*smpcount*K];

		for (int i = 0; i < smpcount; i++) {
			float mu = d_ev[spot * smpcount + i];
			auto jacobian = [=](int j) { return spot_deriv[smpcount*j + i]; };

			mu = max(1e-8f, mu);

			if (sample) {
				float err = sample[smpcount * spot + i] - mu;
				spot_chisq += err * err / mu;
			}

			float inv_mu_c = 1.0f / mu;
			for (int i = 0; i < K; i++) {
				for (int j = i; j < K; j++) {
					const float fi_ij = jacobian(i) * jacobian(j) * inv_mu_c;
					fi[K*i + j] += fi_ij;
				}
			};
		}
		// fill below diagonal
		for (int i = 1; i < K; i++)
			for (int j = 0; j < i; j++)
				fi[K*i + j] = fi[K*j + i];

		float* fi_inv = &d_fi_inv[K * K * spot];
		InvertMatrix(K, fi, &d_P[(K + 1) * spot], fi_inv);

		for (int i = 0; i < K; i++)
			spot_crlb[i] = sqrtf(fi_inv[i * K + i]);

		chisq[spot] = spot_chisq;
	}, 0, stream);
}


//model.ComputeDerivatives([&](int smpIndex, T mu, const T* jacobian) {

__device__ void ComputeAlphaBeta(const float* spot_mu, const float* spot_jac, const float* spot_smp, float* lm_alpha, float* lm_beta, int smpcount, int K)
{
	for (int i = 0; i < K * K; i++)
		lm_alpha[i] = 0.0f;
	for (int i = 0; i < K; i++)
		lm_beta[i] = 0.0f;

	for (int s = 0; s < smpcount; s++) {
		float mu = spot_mu[s];
		float smp = spot_smp[s];
		if (smp < 1e-6f) smp = 1e-6f;

		float mu_c = mu > 1e-6f ? mu : 1e-6f;
		float invmu = 1.0f / mu_c;
		float x_f2 = smp * invmu * invmu;

		for (int i = 0; i < K; i++)
			for (int j = i; j < K; j++)
				lm_alpha[K * i + j] += spot_jac[i * smpcount + s] * spot_jac[j * smpcount + s] * x_f2;

		float beta_factor = 1 - smp * invmu;
		for (int i = 0; i < K; i++) {
			lm_beta[i] -= beta_factor * spot_jac[i * smpcount + s];
		}
	}

	// fill below diagonal
	for (int i = 1; i < K; i++)
		for (int j = 0; j < i; j++)
			lm_alpha[K * i + j] = lm_alpha[K * j + i];

}

void cuEstimator::Estimate(const float * d_sample, const float * d_const, const int* d_roipos, const float * d_initial, 
	float * d_params, float* d_diag, int* iterations, int numspots, float * d_trace, int traceBufLen, hipStream_t stream)
{
	auto db = GetDeviceBuffers(stream, numspots);

	float* d_deriv = db->psf_deriv.data();
	float* d_ev = db->psf_ev.data();
	float* lm_alphabeta = db->lm_alphabeta.data();
	float* lm_lu = db->lm_lu.data();
	int smpcount = SampleCount();
	int K = NumParams();
	ParamLimit* d_limits = this->d_limits.data();

	if (!d_initial)
		return;

	if (d_params != d_initial) {
		ThrowIfCUDAError(hipMemcpyAsync(d_params, d_initial, sizeof(float) * K * numspots, hipMemcpyDeviceToDevice, stream));
	}

	for (int i = 0; i < lmParams.iterations; i++)
	{
		Derivatives(d_deriv, d_ev, d_params, d_const, d_roipos, numspots, stream);

		db->lm_alphabeta.Clear(stream);
		float lambda = lmParams.lambda;

		LaunchKernel(numspots, [=]__device__(int spot) {
			float* lm_alpha = &lm_alphabeta[spot * K * (K + 1)];
			float* lm_beta = &lm_alpha[K * K];
			const float* spot_mu = &d_ev[spot * smpcount];
			const float* spot_jac = &d_deriv[spot * smpcount * K];

			ComputeAlphaBeta(spot_mu, spot_jac, &d_sample[smpcount * spot], lm_alpha, lm_beta, smpcount, K);

			for (int k = 0; k < K; k++) {
				if (lambda > 0.0f) {
					float s = 0.0f; // scale invariant
					for (int j = 0; j < K; j++)
						s += lm_alpha[j * K + k] * lm_alpha[j * K + k];
					lm_alpha[k * K + k] += s * lambda;
				} else 
					lm_alpha[k * K + k] -= lambda; // non scale invariant
			}
		}, 0, stream);

		LaunchKernel(numspots, [=]__device__(int spot) {
			float* lm_alpha = &lm_alphabeta[spot * K * (K + 1)];
			float* lm_beta = &lm_alpha[K * K];
			float* lu = &lm_lu[K * K * spot];
			if (!Cholesky(K, lm_alpha, lu))
				return;
			float* step = lm_alpha; // alpha is not needed anymore at this point
			float* temp = lm_alpha + K;
			if (!SolveCholesky(K, lu, lm_beta, step, temp))
				return;

			for (int k = 0; k < K; k++) {
				float theta = d_params[K * spot + k];
				if (d_trace && i<traceBufLen)
					d_trace[K * traceBufLen * spot + K * i + k] = theta;

				theta += step[k];
				theta = fmax(d_limits[k].min, theta);
				theta = fmin(d_limits[k].max, theta);
				d_params[K * spot + k] = theta;
			}
			if(iterations)
				iterations[spot] = i;
		}, 0, stream);
	}
}

cuEstimatorWrapper::cuEstimatorWrapper(cuEstimator * cudaPSF) : 
	Estimator(cudaPSF->SampleSize(), cudaPSF->NumConstants(), cudaPSF->DiagSize(), cudaPSF->ParamFormat(), cudaPSF->ParamLimits()), psf(cudaPSF)
{}


cuEstimatorWrapper::~cuEstimatorWrapper()
{
	delete psf;
}

void cuEstimatorWrapper::ChiSquareAndCRLB(const float* h_params, const float* h_sample, const float* h_const,
	const int* h_roipos, float* h_crlb, float* h_chisq, int numspots)
{
	DeviceArray<float> d_chisq(numspots);
	DeviceArray<float> d_smp(numspots * SampleCount(), h_sample);
	DeviceArray<float> d_params(numspots*NumParams(), h_params);
	DeviceArray<float> d_crlb(numspots * NumParams(), h_params);
	DeviceArray<float> d_const(numspots*NumConstants(), h_const);
	DeviceArray<int> d_roipos(numspots*SampleIndexDims(), h_roipos);
	psf->ChiSquareAndCRLB(d_params.ptr(), h_sample ? d_smp.ptr() : 0, d_const.ptr(), d_roipos.ptr(), d_crlb.ptr(), d_chisq.ptr(), numspots, 0);
	
	if (h_chisq) d_chisq.CopyToHost(h_chisq);
	if (h_crlb) d_crlb.CopyToHost(h_crlb);
}


void cuEstimatorWrapper::ExpectedValue(float * h_expectedvalue, const float * h_theta, const float* h_const, const int* spot_pos, int numspots)
{
	DeviceArray<float> d_ev(numspots*SampleCount());
	DeviceArray<float> d_params(numspots*NumParams(), h_theta);
	DeviceArray<float> d_const(numspots*NumConstants(), h_const);
	DeviceArray<int> d_roipos(numspots*SampleIndexDims(), spot_pos);
	psf->ExpectedValue(d_ev.ptr(), d_params.ptr(), d_const.ptr(), d_roipos.ptr(), numspots, 0);
	d_ev.CopyToHost(h_expectedvalue);
}

void cuEstimatorWrapper::Derivatives(float * h_deriv, float * h_expectedvalue, const float * h_theta, const float* h_const, const int* spot_pos, int numspots)
{
	DeviceArray<float> d_params(numspots*NumParams(), h_theta);
	DeviceArray<float> d_ev(numspots*SampleCount());
	DeviceArray<float> d_deriv(numspots*NumParams()*psf->SampleCount());
	DeviceArray<float> d_const(numspots*NumConstants(), h_const);
	DeviceArray<int> d_roipos(numspots*SampleIndexDims(), spot_pos);
	psf->Derivatives(d_deriv.ptr(), d_ev.ptr(), d_params.ptr(), d_const.ptr(), d_roipos.ptr(), numspots, 0);
	d_ev.CopyToHost(h_expectedvalue);
	d_deriv.CopyToHost(h_deriv);
}

void cuEstimatorWrapper::Estimate(const float * h_sample, const float* h_const, const int* spot_pos, const float * h_initial, 
	float * h_theta, float * h_diag, int* h_iterations, int numspots, float* h_trace, int traceBufLen)
{
	DeviceArray<float> d_smp(numspots*SampleCount(), h_sample);
	DeviceArray<float> d_initial(h_initial ? numspots * NumParams() : 0, h_initial);
	DeviceArray<float> d_params(numspots*NumParams());
	DeviceArray<float> d_diag(numspots*DiagSize());
	DeviceArray<float> d_trace(numspots*traceBufLen*NumParams());
	DeviceArray<float> d_const(numspots*NumConstants(), h_const);
	DeviceArray<int> d_roipos(numspots*SampleIndexDims(), spot_pos);
	DeviceArray<int> d_iterations(numspots);
	psf->Estimate(d_smp.ptr(), d_const.ptr(), d_roipos.ptr(), d_initial.ptr(), d_params.ptr(), 
		d_diag.ptr(), d_iterations.ptr(), numspots, d_trace.ptr(), traceBufLen, 0);
	d_params.CopyToHost(h_theta);
	if (h_trace) d_trace.CopyToHost(h_trace);
	if (h_diag) d_diag.CopyToHost(h_diag);
	if (h_iterations) d_iterations.CopyToHost(h_iterations);
}

void cuEstimatorWrapper::SetLMParams(LMParams p)
{
	psf->SetLMParams(p);
}

LMParams cuEstimatorWrapper::GetLMParams()
{
	return psf->GetLMParams();
}

CenterOfMassEstimator::CenterOfMassEstimator(int roisize) : roisize(roisize),
	cuEstimator({ roisize,roisize }, 0, 0, "x,y,I,bg",
		{ {0.0f,roisize - 1.0f},{0.0f,roisize - 1.0f},{0.0f,1e9f},{0.0f,0.0f } })
{}

void CenterOfMassEstimator::Estimate(const float* d_sample, const float *d_const, const int* d_roipos, const float* d_initial, float* d_params, float* d_diag, int* d_iterations,
	int numspots, float * d_trace, int traceBufLen, hipStream_t stream)
{
	int numThreads = 1024;
	int roisize = SampleSize()[0];
	Vector4f* estim = (Vector4f*)d_params;
	LaunchKernel(numspots, [=]__device__(int i) {
		const float* smp = &d_sample[i*roisize*roisize];
		auto com = ComputeCOM(smp, { roisize,roisize });
		estim[i] = { com[0],com[1],com[2],0.0f };
		d_iterations[i] = 0;
	}, 0, stream, numThreads);
}

PhasorEstimator::PhasorEstimator(int roisize) : roisize(roisize), 
	cuEstimator({ roisize,roisize }, 0, 0, "x,y,I,bg", {
		{0.0f, roisize - 1.0f },
		{0.0f, roisize - 1.0f },
		{0.0f, 1e9f },
		{1e-6f, 1e9f },
	})
{}

void PhasorEstimator::Estimate(const float* d_sample, const float *d_const, const int* d_roipos, const float* d_initial, 
	float* d_params, float* d_diag, int* d_iterations, int numspots, float * d_trace, int traceBufLen, hipStream_t stream)
{
	int roisize = SampleSize()[0];
	Vector4f* estim = (Vector4f*)d_params;
	LaunchKernel(numspots, [=]__device__(int i) {
		const float* smp = &d_sample[i*roisize*roisize];

		Vector3f e = ComputePhasorEstim(smp, roisize,roisize );
		estim[i] = { e[0],e[1],e[2],0.0f };

		d_iterations[i] = 0;
	}, 0, stream, 1024);
}


CDLL_EXPORT Estimator * CreateCenterOfMassEstimator(int roisize, Context* ctx)
{
	auto* p = new cuEstimatorWrapper(new CenterOfMassEstimator(roisize));
	if (ctx) p->SetContext(ctx);
	return p;
}

CDLL_EXPORT Estimator* CreatePhasorEstimator(int roisize, Context* ctx)
{
	auto* p = new cuEstimatorWrapper(new PhasorEstimator(roisize));
	if (ctx) p->SetContext(ctx);
	return p;
}
